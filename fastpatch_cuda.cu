#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include "types.h"

#ifndef THREAD_NUM
#define THREAD_NUM 32
#endif  // THREAD_NUM

namespace fastpatch {
namespace {

#define CHECK_RUNTIME_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

}  // namespace

// function: (const) torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> tensor
// pass args: tensor.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>()
__global__ void feat_forward_kernel(int N, int maxsize, int Cin,
    const Neighbor_t*  __restrict__ nn_list, const float* __restrict__  feat_data,
    float* __restrict__  featpatch_data) {

        int u = blockIdx.x;  // bi
        int N_RW = blockDim.x;
        int PATCH_STRIDE = maxsize * Cin;

        // TODO(BUG): alling a __host__ from a __global__ function is not allowed
        int Ns = torch::size(nn_list[u], 0);
        // TODO(BUG) host code
        int* nn = nn_list[u].data_ptr<int>(); // Ns

        int ti = threadIdx.x;
        
        for (int i = ti; i < Ns; i += N_RW) {
            int v = nn[i];
            hipMemcpyAsync(featpatch_data + u * PATCH_STRIDE + i * Cin, feat_data + v * Cin,
                Cin * sizeof(float), hipMemcpyDeviceToDevice);
        }
}

__global__ void feat_backward_kernel(int N, int maxsize, int Cin,
    const Neighbor_t*  __restrict__ grad_nn_list, const float* __restrict__  grad_patchfeat,
    float* __restrict__  grad_feat) {

        int u = blockIdx.x;  // bi
        int N_RW = blockDim.x;
        int PATCH_STRIDE = maxsize * Cin;

        // TODO(BUG) host code
        int Ns = torch::size(grad_nn_list[u], 0);
        // TODO(BUG) host code
        int* grad_nn =  grad_nn_list[u].data_ptr<int>();  // Ns x 2

        int ti = threadIdx.x;
        int tj = threadIdx.y;
        
        for (int i = ti; i < Ns; i += N_RW) {
            int v = grad_nn[i * 2];
            int offset = grad_nn[i * 2 + 1];
            grad_feat[u * Cin + tj] += grad_patchfeat[v * PATCH_STRIDE + offset * Cin + tj];
        }
}


torch::Tensor feat_forward(torch::Tensor feat, NnList_t& nn_list, int maxsize) {
    // Not Implemented
    CHECK_CUDA(feat);
    
    int N = nn_list.size();
    int Cin = torch::size(feat, 1);

    torch::Tensor patchfeat = torch::zeros({N, maxsize, Cin, 1}, feat.options());

    const dim3 block(THREAD_NUM);
    const dim3 grid(N);
    feat_forward_kernel<<<grid, block>>>(
        N, maxsize, Cin, nn_list.data(),
        feat.data_ptr<float>(), patchfeat.data_ptr<float>());
    
    CHECK_RUNTIME_ERROR(hipPeekAtLastError());
    return patchfeat;
}


torch::Tensor feat_backward(torch::Tensor grad_patchfeat, GradNnList_t& grad_nn_list, int maxsize) {
    CHECK_CUDA(grad_patchfeat);

    int N = grad_nn_list.size();
    int Cin = torch::size(grad_patchfeat, 2);  // N x maxsize x Cin x 1

    torch::Tensor grad_feat = torch::zeros({N, Cin, 1}, grad_patchfeat.options());

    const dim3 block(THREAD_NUM, Cin);
    const dim3 grid(N);
    feat_backward_kernel<<<grid, block>>>(
        N, maxsize, Cin, grad_nn_list.data(),
        grad_patchfeat.data_ptr<float>(), grad_feat.data_ptr<float>());
    
    CHECK_RUNTIME_ERROR(hipPeekAtLastError());
    return grad_feat;
}


__global__ void get_selection_mat_kernel(
    int maxsize, int S, const Weight_t* __restrict__ nw_list, float* __restrict__ select_mat) {
        int bi = blockIdx.x;
        int STRIDE = maxsize * S;

        // TODO(BUG) host code
        int Ns = torch::size(nw_list[bi], 0);
        // TODO(BUG) host code
        float* nw = nw_list[bi].data_ptr<float>();

        hipMemcpyAsync(select_mat + bi * STRIDE, nw,
            Ns * S * sizeof(float), hipMemcpyDeviceToDevice);
}


torch::Tensor get_selection_mat(int S, NnList_t& nn_list, NwList_t& nw_list, int maxsize) {
    CHECK_CUDA(nw_list[0]);
    int N = nw_list.size();

    torch::Tensor select_mat = torch::zeros({N, maxsize, S}, nw_list[0].options());

    const dim3 block(1);
    const dim3 grid(N);
    get_selection_mat_kernel<<<grid, block>>>(
        maxsize, S, nw_list.data(), select_mat.data_ptr<float>());
    
    CHECK_RUNTIME_ERROR(hipPeekAtLastError());
    return select_mat;
}


GradNnList_t grad_nn_list(NnList_t& nn_list) {
    int N = nn_list.size();
    std::vector<std::vector<int>> grad_nn_v(N);
    std::vector<std::vector<int>> grad_nn_offset(N);

    for (int u = 0; u < N; ++u) {
        int Ns = torch::size(nn_list[u], 0);
        int* nn = nn_list[u].data_ptr<int>();
        for (int j = 0; j < Ns; ++j) {
            grad_nn_v[nn[j]].push_back(u);
            grad_nn_offset[nn[j]].push_back(j);
        }
    }
    GradNnList_t grad_nn_list;
    grad_nn_list.reserve(N);
    for (int u = 0; u < N; ++u) {
        int Ns = grad_nn_v[u].size();
        torch::Tensor grad_nn = torch::zeros({Ns, 2}, nn_list[0].options());
        grad_nn_list.push_back(grad_nn);
        
        int* grad_nn_ptr = grad_nn.data_ptr<int>();
        for (int i = 0; i < grad_nn_v.size(); ++i) {
            grad_nn_ptr[i * 2] = grad_nn_v[u][i];
            grad_nn_ptr[i * 2 + 1] = grad_nn_offset[u][i];
        }
    }
    return grad_nn_list;
}

}  // namespace fastpatch
